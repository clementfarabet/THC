#include "hip/hip_runtime.h"
#include "THCTensor.h"
#include "THCGeneral.h"
#include "THGeneral.h"

static void THCudaTensor_computesz(THCudaTensor *self, long **sz_, long **st_)
{
  long *sz, *st, *szh;
  int i;
  
  THCudaCheck(hipMalloc(&sz, sizeof(long)*self->nDimension));
  THCudaCheck(hipMalloc(&st, sizeof(long)*self->nDimension));
  szh = (long*)THAlloc(sizeof(long)*self->nDimension);

  for(i = self->nDimension-1; i >= 0; i--)
  {
    if(i == self->nDimension-1)
      szh[i] = 1;
    else
      szh[i] = szh[i+1]*self->size[i+1];
  }

  THCudaCheck(hipMemcpy(sz, szh, self->nDimension * sizeof(long), hipMemcpyHostToDevice));
  THCudaCheck(hipMemcpy(st, self->stride, self->nDimension * sizeof(long), hipMemcpyHostToDevice));
  THFree(szh);

  *sz_ = sz;
  *st_ = st;
}

__global__ void THCudaTensor_kernel_copy(float *dst, 
                                         long *dst_sz, long *dst_st, int dst_dim,
                                         float *src,
                                         long *src_sz, long *src_st, int src_dim,
                                         long n_elem)
{
  long k = (((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x) + threadIdx.x;
  
  if(k < n_elem)
  {
    long dst_idx = 0;
    long dst_rest = k;
    for(int dim = 0; dim < dst_dim; dim++)
    {
      dst_idx += (dst_rest/dst_sz[dim])*dst_st[dim];
      dst_rest = dst_rest % dst_sz[dim];
    }

    long src_idx = 0;
    long src_rest = k;
    for(int dim = 0; dim < src_dim; dim++)
    {
      src_idx += (src_rest/src_sz[dim])*src_st[dim];
      src_rest = src_rest % src_sz[dim];
    }

    dst[dst_idx] = src[src_idx];
  }
}

void THCudaTensor_copy(THCudaTensor *self, THCudaTensor *src)
{
  THArgCheck(THCudaTensor_nElement(self) == THCudaTensor_nElement(src), 2, "sizes do not match"); 

  if(THCudaTensor_isContiguous(self) && THCudaTensor_isContiguous(src))
    THCudaCheck(hipMemcpy(self->storage->data + self->storageOffset, src->storage->data + src->storageOffset, THCudaTensor_nElement(src) * sizeof(float), hipMemcpyDeviceToDevice));
  else
  {    
    long *d_self_sz, *d_self_st, *d_src_sz, *d_src_st;
    long size = THCudaTensor_nElement(self);

    THCudaTensor_computesz(self, &d_self_sz, &d_self_st);
    THCudaTensor_computesz(src, &d_src_sz, &d_src_st);

    int nBlockPerRow, nBlockPerColumn, nThreadPerBlock;
    THCudaGetGridSize(&nBlockPerRow, &nBlockPerColumn, &nThreadPerBlock, size);    
    dim3 threads(nThreadPerBlock);
    dim3 grid(nBlockPerRow, nBlockPerColumn);

    THCudaTensor_kernel_copy<<<grid, threads>>>(THCudaTensor_data(self), 
                                                d_self_sz, d_self_st, self->nDimension,
                                                THCudaTensor_data(src),
                                                d_src_sz, d_src_st, src->nDimension,
                                                size);

    hipError_t errcode = hipGetLastError();
    if(errcode != hipSuccess)
      THError(hipGetErrorString(errcode));

    hipDeviceSynchronize();

    THCudaCheck(hipFree(d_self_sz));
    THCudaCheck(hipFree(d_self_st));
    THCudaCheck(hipFree(d_src_sz));
    THCudaCheck(hipFree(d_src_st));
  }
}
