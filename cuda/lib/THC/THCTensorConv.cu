#include "hip/hip_runtime.h"
#include "THCTensorConv.h"
#include "THCGeneral.h"

/*
 * Description:
 *   This code provides convolutions and xcorrelations that are API compatible with
 *   the ones in THLabConv.
 *
 * History:
 *   Sept 11, 2011, 11:59PM  -  Clement Farabet  -  Optimized RevConv by a good x2
 *   July 22, 2011, 8:38PM   -  Clement Farabet  -  All Valid/Full/XCORR/CONV implemented
 *   July 22, 2011, 4:00PM   -  Clement Farabet  -  Rewrote for loop to insure memory coalescing
 *   July 21, 2011, 11:21PM  -  Clement Farabet  -  Creation, based conv2d routine
 */

#define CUDA_SHARED_MEM_SIZE (4*1024-32) // this is given by nVidia: max shared mem per block

/*
 * Description:
 *   base conv2D routine: 3D input, 3D output, 4D kernel
 *
 *   - all chunks of data should be contiguous
 *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
 *   - the templated kernel size is useful to generate code that's 2x faster
 *     but can be set to 0 to allow arbitrary kernel sizes
 */
template <bool swapkernel, int T_kernel_h, int T_kernel_w>
  __global__ void conv2generic(float *input, float *kernel, float *output,
                               int input_n, int input_h, int input_w,
                               int kernel_n, int kernel_h, int kernel_w,
                               int stride_h, int stride_w)
{
  // output dimensions
  int output_h = (input_h - kernel_h) / stride_h + 1;
  int output_w = (input_w - kernel_w) / stride_w + 1;

  // xcorr or conv
  int koffset = swapkernel ? kernel_w*kernel_h-1 : 0;

  // generate offsets according to block/thread ids
  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y;

  int oo_start = blockIdx.x;
  int oo_end = oo_start+1;

  int ii_start = 0;
  int ii_end = input_n;

  // nb threads, unique thread id
  int tid = blockDim.x * threadIdx.y + threadIdx.x;
  int nthreads = blockDim.x * blockDim.y;

  // iterators
  int oo, ii, xx, yy, kx, ky, kk;

  // do the kernels fit in shared mem ?
  if (input_n*kernel_w*kernel_h <= CUDA_SHARED_MEM_SIZE) {

    // put the kernel in shared memory
    __shared__ float shared_kernel[CUDA_SHARED_MEM_SIZE];

    // first thread of each block does the copy
    for (kk = tid; kk < kernel_w*kernel_h*input_n; kk += nthreads) {
      shared_kernel[kk] = kernel[input_n*kernel_w*kernel_h*blockIdx.x + kk];
    }
    __syncthreads();

    // templated kernel size
    if ((T_kernel_w > 0) && (T_kernel_h > 0)) {
      // unrolled convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for(ii = ii_start; ii < ii_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = shared_kernel + ii * kernel_w * kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
#pragma unroll
                for(ky = 0; ky < T_kernel_h; ky++) {
#pragma unroll
                  for(kx = 0; kx < T_kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    } else {
      // default convolution loop
      for(oo = oo_start; oo < oo_end; oo++) {
        for(ii = ii_start; ii < ii_end; ii++) {
          for(yy = yy_start; yy < yy_end; yy+=yy_step) {
            for(xx = xx_start; xx < xx_end; xx+=xx_step) {
              // Dot product in two dimensions... (between input image and the mask)
              float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
              float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
              float *kernel_p = shared_kernel + ii * kernel_w * kernel_h + koffset;
              float sum = 0;
              if (swapkernel) {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p--);
                  }
                  input_p += input_w;
                }
              } else {
                for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                  for(kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx]*(*kernel_p++);
                  }
                  input_p += input_w;
                }
              }
              *output_p += sum;
            }
          }
        }
      }
    }

  } else { // not enough shared mem for kernels, simply stream them

    // convolution loop
    for(oo = oo_start; oo < oo_end; oo++) {
      for(ii = ii_start; ii < ii_end; ii++) {
        for(yy = yy_start; yy < yy_end; yy+=yy_step) {
          for(xx = xx_start; xx < xx_end; xx+=xx_step) {
            // Dot product in two dimensions... (between input image and the mask)
            float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
            float *output_p = output + oo*output_h*output_w + yy*output_w + xx;
            float *kernel_p = kernel + (oo * input_n + ii) * kernel_w * kernel_h + koffset;
            float sum = 0;
            if (swapkernel) {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p--);
                }
                input_p += input_w;
              }
            } else {
              for(ky = 0; ky < kernel_h; ky++) {
#pragma unroll 5
                for(kx = 0; kx < kernel_w; kx++) {
                  sum += input_p[kx]*(*kernel_p++);
                }
                input_p += input_w;
              }
            }
            *output_p += sum;
          }
        }
      }
    }
  }
}

/*
 * Description:
 *   base conv2D routine with reversed stride: 3D input, 4D output, 3D kernel
 *   this is useful for computing gradients with respect to kernels, where:
 *   input=input, kernel=gradOutput, output=gradWeight
 *
 *   - all chunks of data should be contiguous
 *   - the swapkernel flag can be used to generate a conv2 instead of xcorr2
 */
__global__ void conv2genericrev(float *input, float *kernel, float *output,
                                int input_n, int input_h, int input_w,
                                int kernel_n, int kernel_h, int kernel_w,
                                float alpha, int stride_h, int stride_w)
{
  // output dimensions
  int output_h = input_h - (kernel_h - 1) * stride_h;
  int output_w = input_w - (kernel_w - 1) * stride_w;

  // this thread only processes one output, defined by the block Ids
  int kk = blockIdx.x;
  int ii = blockIdx.y;

  // thread ID
  int tid = threadIdx.x;
  int nthreads = blockDim.x;

  // one thread only sees one output
  output = output + (kk * input_n + ii) * output_h*output_w;

  // put the output in shared memory
  __shared__ float shared_output[CUDA_SHARED_MEM_SIZE];

  // generate tid outputs in shared memory
  float *output_s = shared_output + tid*output_w*output_h;

  // convolution loop
  int xx, yy, kx, ky;
  yy = threadIdx.y;
  float *output_p = output_s + yy * output_w;
  for(xx=0; xx<output_w; xx++) {
    // Dot product in two dimensions... (between input image and kernel)
    float *input_p = input + ii*input_h*input_w + yy*stride_h*input_w + xx*stride_w;
    float *kernel_p = kernel + kk*kernel_w*kernel_h;
    float sum = 0;
    for(ky=0; ky<kernel_h; ky++) {
      for(kx=tid; kx<kernel_w; kx+=nthreads) {
        sum += input_p[kx]*kernel_p[kx];
      }
      input_p += input_w;
      kernel_p += kernel_w;
    }
    *(output_p++) = sum;
  }
  __syncthreads();

  // reduce and write back
  if (yy == 0) {
    // reduce outputs
    for (int k=1; k<nthreads; k++) {
      for (int i=tid; i<output_w*output_h; i+=nthreads) {
        shared_output[i] += shared_output[k*output_h*output_w + i];
      }
    }
    __syncthreads();

    // add existing output, and write back
    for (int i=tid; i<output_w*output_h; i+=nthreads) {
      output[i] += alpha*shared_output[i];
    }
  }
}

/*
 * API-compatible with THRealTensor_conv2Dmv
 * 3D input, 4D kernel, 3D output
 * matrix vector product like: y <- Ax + beta*y
 */
TH_API void THCudaTensor_conv2Dmv(THCudaTensor *output, float beta, THCudaTensor *input,
                                  THCudaTensor *kernel, long srow, long scol, const char *type)
{
  long nInputPlane, nInputRows, nInputCols;
  long nKernelRows, nKernelCols;
  long nOutputPlane, nOutputRows, nOutputCols;

  THArgCheck(kernel->nDimension == 4 , 4, "kernel: 4D Tensor expected");
  THArgCheck(srow >= 1, 5, "Stride should be a positive integer");
  THArgCheck(scol >= 1, 6, "Stride should be a positive integer");
  THArgCheck(type[0] == 'v' || type[0] == 'f', 7, "type of convolution can 'v' or 'f'");
  THArgCheck(type[1] == 'c' || type[1] == 'x', 7, "type of convolution can 'x' or 'c'");

  input = THCudaTensor_newContiguous(input);
  kernel = THCudaTensor_newContiguous(kernel);

  nInputPlane = input->size[0];
  nInputRows  = input->size[1];
  nInputCols  = input->size[2];

  nKernelRows  = kernel->size[2];
  nKernelCols  = kernel->size[3];
  nOutputPlane = kernel->size[0];
  THArgCheck(kernel->size[1] == nInputPlane, 2, "invalid number of input planes");

  THArgCheck( (nInputRows >= nKernelRows && nInputCols >= nKernelCols) || *type == 'f', 2,
              "conv2Dmv : Input image is smaller than kernel");

  if (*type == 'f') {
    // output dims
    nOutputRows = (nInputRows - 1) * srow + nKernelRows;
    nOutputCols = (nInputCols - 1) * scol + nKernelCols;

    // use temp buffer
    static THCudaTensor *inputP;
    static int firstcall = 1;
    if (firstcall) {
      inputP = THCudaTensor_new();
      firstcall = 0;
    }

    // create a zero-padded input
    long nInputRowsPadded = (nOutputRows - 1) * srow + nKernelRows;
    long nInputColsPadded = (nOutputCols - 1) * scol + nKernelCols;
    THCudaTensor_resize3d(inputP, nInputPlane, nInputRowsPadded, nInputColsPadded);
    THCudaTensor_zero(inputP);

    THCudaTensor *centered = THCudaTensor_new();
    THCudaTensor_narrow(centered, inputP, 2, nKernelCols-1, nInputCols);
    THCudaTensor_narrow(centered, NULL, 1, nKernelRows-1, nInputRows);
    THCudaTensor_copy(centered, input);
    THCudaTensor_free(centered);

    // remap input to newly created tensor
    THCudaTensor_free(input);
    input = inputP;
    nInputRows = nInputRowsPadded;
    nInputCols = nInputColsPadded;

  } else { // 'v'
    // output dims
    nOutputRows = (nInputRows - nKernelRows) / srow + 1;
    nOutputCols = (nInputCols - nKernelCols) / scol + 1;
  }

  long nelem = THCudaTensor_nElement(output);
  THCudaTensor_resize3d(output, nOutputPlane, nOutputRows, nOutputCols);

  if (beta == 0 || nelem != THCudaTensor_nElement(output)) {
    THCudaTensor_zero(output);
  } else if (beta != 1) {
    THCudaTensor_mul(output, beta);
  }

  float *input_data = THCudaTensor_data(input);
  float *weight_data = THCudaTensor_data(kernel);
  float *output_data = THCudaTensor_data(output);

  // cuda blocks & threads:
  dim3 blocks(nOutputPlane);
  dim3 threads(32, 8);

  // sync any previous kernel exec
  hipDeviceSynchronize();

  // convolution: xcorr2 or conv2
  if (type[1] == 'x') {
    if ((nKernelCols == 3) && (nKernelRows == 3))
      conv2generic <false, 3, 3> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
    else if ((nKernelCols == 5) && (nKernelRows == 5))
      conv2generic <false, 5, 5> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
    else if ((nKernelCols == 7) && (nKernelRows == 7))
      conv2generic <false, 7, 7> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
    else if ((nKernelCols == 9) && (nKernelRows == 9))
      conv2generic <false, 9, 9> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
    else if ((nKernelCols == 11) && (nKernelRows == 11))
      conv2generic <false, 11, 11> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                          nInputPlane, nInputRows, nInputCols,
                                                          nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                          srow, scol);
    else if ((nKernelCols == 13) && (nKernelRows == 13))
      conv2generic <false, 13, 13> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                          nInputPlane, nInputRows, nInputCols,
                                                          nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                          srow, scol);
    else if ((nKernelCols == 4) && (nKernelRows == 4))
      conv2generic <false, 4, 4> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
    else if ((nKernelCols == 6) && (nKernelRows == 6))
      conv2generic <false, 6, 6> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
    else if ((nKernelCols == 8) && (nKernelRows == 8))
      conv2generic <false, 8, 8> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
    else if ((nKernelCols == 10) && (nKernelRows == 10))
      conv2generic <false, 10, 10> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                          nInputPlane, nInputRows, nInputCols,
                                                          nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                          srow, scol);
    else if ((nKernelCols == 12) && (nKernelRows == 12))
      conv2generic <false, 12, 12> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                          nInputPlane, nInputRows, nInputCols,
                                                          nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                          srow, scol);
    else
      conv2generic <false, 0 , 0> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                         nInputPlane, nInputRows, nInputCols,
                                                         nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                         srow, scol);
  } else { // 'c'
    if ((nKernelCols == 3) && (nKernelRows == 3))
      conv2generic <true, 3, 3> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 5) && (nKernelRows == 5))
      conv2generic <true, 5, 5> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 7) && (nKernelRows == 7))
      conv2generic <true, 7, 7> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 9) && (nKernelRows == 9))
      conv2generic <true, 9, 9> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 11) && (nKernelRows == 11))
      conv2generic <true, 11, 11> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                         nInputPlane, nInputRows, nInputCols,
                                                         nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                         srow, scol);
    else if ((nKernelCols == 13) && (nKernelRows == 13))
      conv2generic <true, 13, 13> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                         nInputPlane, nInputRows, nInputCols,
                                                         nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                         srow, scol);
    else if ((nKernelCols == 2) && (nKernelRows == 2))
      conv2generic <true, 2, 2> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 4) && (nKernelRows == 4))
      conv2generic <true, 4, 4> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 6) && (nKernelRows == 6))
      conv2generic <true, 6, 6> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 8) && (nKernelRows == 8))
      conv2generic <true, 8, 8> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                       nInputPlane, nInputRows, nInputCols,
                                                       nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                       srow, scol);
    else if ((nKernelCols == 10) && (nKernelRows == 10))
      conv2generic <true, 10, 10> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                         nInputPlane, nInputRows, nInputCols,
                                                         nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                         srow, scol);
    else if ((nKernelCols == 12) && (nKernelRows == 12))
      conv2generic <true, 12, 12> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                         nInputPlane, nInputRows, nInputCols,
                                                         nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                         srow, scol);
    else
      conv2generic <true, 0 , 0> <<<blocks, threads>>> (input_data, weight_data, output_data,
                                                        nInputPlane, nInputRows, nInputCols,
                                                        nOutputPlane*nInputPlane, nKernelRows, nKernelCols,
                                                        srow, scol);
  }

  // sync & clean
  hipDeviceSynchronize();
  if (*type != 'f') THCudaTensor_free(input);
  THCudaTensor_free(kernel);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in conv2Dmv: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}

/*
 * API-compatible with THRealTensor_conv2DRevger
 * 3D input, 3D kernel, 4D output
 * like rank1 update
 * A <- xx' + beta*A
 * for sr,sc=1 this is equivalent to xcorr2Dger, but otherwise it is useful for
 * calculating derivatives wrt a kernel that is applied with stride sr,sc != 1
 */
TH_API void THCudaTensor_conv2DRevger(THCudaTensor *output, float beta, float alpha,
                                      THCudaTensor *input, THCudaTensor *kernel,
                                      long srow, long scol)
{
  long nInputPlane, nInputRows, nInputCols;
  long nKernelPlane, nKernelRows, nKernelCols;
  long nOutputRows, nOutputCols;

  THArgCheck(input->nDimension == 3 , 3, "input: 3D Tensor expected");
  THArgCheck(kernel->nDimension == 3 , 4, "kernel: 3D Tensor expected");
  THArgCheck(srow >= 1, 5, "Stride should be a positive integer");
  THArgCheck(scol >= 1, 6, "Stride should be a positive integer");

  input = THCudaTensor_newContiguous(input);
  kernel = THCudaTensor_newContiguous(kernel);

  nInputPlane = input->size[0];
  nInputRows  = input->size[1];
  nInputCols  = input->size[2];

  nKernelPlane = kernel->size[0];
  nKernelRows = kernel->size[1];
  nKernelCols = kernel->size[2];

  THArgCheck(nInputRows >= nKernelRows && nInputCols >= nKernelCols , 2,
             "conv2DRevger : Input image is smaller than kernel");

  nOutputRows = nInputRows - (nKernelRows - 1) * srow;
  nOutputCols = nInputCols - (nKernelCols - 1) * scol;

  long nelem = THCudaTensor_nElement(output);
  THCudaTensor_resize4d(output, nKernelPlane, nInputPlane, nOutputRows, nOutputCols);

  if (nelem == 0 || beta == 0 || nelem != THCudaTensor_nElement(output)) {
    THCudaTensor_zero(output);
  } else if (beta != 1) {
    THCudaTensor_mul(output, beta);
  }

  float *input_data = THCudaTensor_data(input);
  float *kernel_data = THCudaTensor_data(kernel);
  float *output_data = THCudaTensor_data(output);

  // auto compute nb of blocks and threads
  dim3 blocks(nKernelPlane, nInputPlane);
  dim3 threads(128/nOutputRows, nOutputRows);

  // sync previous jobs
  hipDeviceSynchronize();

  // compute rev conv
  conv2genericrev <<<blocks, threads>>> (input_data, kernel_data, output_data,
                                         nInputPlane, nInputRows, nInputCols,
                                         nKernelPlane, nKernelRows, nKernelCols,
                                         alpha, srow, scol);

  // sync & clean
  hipDeviceSynchronize();
  THCudaTensor_free(input);
  THCudaTensor_free(kernel);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in conv2DRevger: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}
